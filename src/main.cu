#include "hip/hip_runtime.h"
#include <iostream>

__device__ int mod(int a, int b)
{
    return (a + b) % b;
}

__device__ char nextState(char slice[9])
{
    if (slice[4] == RED_DYING || slice[4] == GREEN_DYING || slice[4] == BLUE_DYING)
        return DEAD;
    if (slice[4] == RED_ALIVE)
        return RED_DYING;
    if (slice[4] == GREEN_ALIVE)
        return GREEN_DYING;
    if (slice[4] == BLUE_ALIVE)
        return BLUE_DYING;
    
    int i, red, green, blue;
    red = green = blue = 0;

    for (i = 0; i < 9; i++)
    {
        if (i == 4)
            continue;
        else if (slice[i] == RED_ALIVE)
            red++;
        else if (slice[i] == GREEN_ALIVE)
            green++;
        else if (slice[i] == BLUE_ALIVE)
            blue++;
    }

    if (red + green + blue == 4)
    {
        if (red >= 2)
            return RED_ALIVE;
        if (green >= 2)
            return GREEN_ALIVE;
        if (blue >= 2)
            return BLUE_ALIVE;
    }

    return DEAD;
}

__global__ void nextGeneration(char oldBoard[HEIGHT][WIDTH], char newBoard[HEIGHT][WIDTH])
{
    int i, j;
    char newBoard[HEIGHT][WIDTH], slice[9];

    for (i = 0; i < HEIGHT; i++)
    {
        for (j = 0; j < WIDTH; j++)
        {
            slice[0] = oldBoard[mod(i - 1, HEIGHT)][mod(j - 1, WIDTH)];
            slice[1] = oldBoard[mod(i - 1, HEIGHT)][j];
            slice[2] = oldBoard[mod(i - 1, HEIGHT)][mod(j + 1, WIDTH)];
            slice[3] = oldBoard[i]                 [mod(j - 1, WIDTH)];
            slice[4] = oldBoard[i]                 [j];
            slice[5] = oldBoard[i]                 [mod(j + 1, WIDTH)];
            slice[6] = oldBoard[mod(i + 1, HEIGHT)][mod(j - 1, WIDTH)];
            slice[7] = oldBoard[mod(i + 1, HEIGHT)][j];
            slice[8] = oldBoard[mod(i + 1, HEIGHT)][mod(j + 1, WIDTH)];

            newBoard[i][j] = nextState(slice);
        }
    }

    memcpy(oldBoard, newBoard, sizeof(char) * WIDTH * HEIGHT);
}

int main (void) {
    cout << "Hello World" << endl;
    return 0;
}