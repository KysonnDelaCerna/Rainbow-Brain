#include "hip/hip_runtime.h"
#include <iostream>
extern "C" {
#include "include/gameOfLife.h"
}

using namespace std;

__device__ int mod(int a, int b)
{
    return (a + b) % b;
}

__device__ char nextState(char slice[9])
{
    if (slice[4] == RED_DYING || slice[4] == GREEN_DYING || slice[4] == BLUE_DYING)
        return DEAD;
    if (slice[4] == RED_ALIVE)
        return RED_DYING;
    if (slice[4] == GREEN_ALIVE)
        return GREEN_DYING;
    if (slice[4] == BLUE_ALIVE)
        return BLUE_DYING;
    
    int i, red, green, blue;
    red = green = blue = 0;

    for (i = 0; i < 9; i++)
    {
        if (i == 4)
            continue;
        else if (slice[i] == RED_ALIVE)
            red++;
        else if (slice[i] == GREEN_ALIVE)
            green++;
        else if (slice[i] == BLUE_ALIVE)
            blue++;
    }

    if (red + green + blue == 4)
    {
        if (red >= 2)
            return RED_ALIVE;
        if (green >= 2)
            return GREEN_ALIVE;
        if (blue >= 2)
            return BLUE_ALIVE;
    }

    return DEAD;
}

__global__ void nextGeneration(char *oldBoard, char *newBoard)
{
    int i, x, y, left, right, up, down, stride = blockDim.x * gridDim.x;
    char slice[9];

    for (i = blockIdx.x * blockDim.x + threadIdx.x; i < WIDTH * HEIGHT; i += stride)
    {
        x = i % WIDTH;
        y = i - x;
        left = mod(x - 1, WIDTH);
        right = mod(x + 1, WIDTH);
        up = mod(y - WIDTH, WIDTH * HEIGHT);
        down = mod(y + WIDTH, WIDTH * HEIGHT);

        slice[0] = oldBoard[left + up];
        slice[1] = oldBoard[x + up];
        slice[2] = oldBoard[right + up];
        slice[3] = oldBoard[left + y];
        slice[4] = oldBoard[i];
        slice[5] = oldBoard[right + y];
        slice[6] = oldBoard[left + down];
        slice[7] = oldBoard[x + down];
        slice[8] = oldBoard[right + down];

        newBoard[i] = nextState(slice);
    }
}

void printBoard(char *board) {
    int i, j;

    for (i = 0; i < 16; i++) {
        for (j = 0; j < 16; j++) {
            switch(board[i * WIDTH + j])
            {
                case RED_ALIVE: cout << "R"; break;
                case RED_DYING: cout << "r"; break;
                case GREEN_ALIVE: cout << "G"; break;
                case GREEN_DYING: cout << "g"; break;
                case BLUE_ALIVE: cout << "B"; break;
                case BLUE_DYING: cout << "b"; break;
                case DEAD: cout << " "; break;
            }
        }
        cout << endl;
    }
    cout << endl;
}

int main (void) {
    char *oldBoard, *newBoard;
    hipMallocManaged(&oldBoard, WIDTH * HEIGHT * sizeof(char));
    hipMallocManaged(&newBoard, WIDTH * HEIGHT * sizeof(char));

    seedRandom();
    initiateRandomBoardP(oldBoard);

    printBoard(oldBoard);

    int blockSize = 256;
    int numBlocks = (WIDTH * HEIGHT + blockSize - 1) / blockSize;

    nextGeneration<<<numBlocks, blockSize>>>(oldBoard, newBoard);
    hipDeviceSynchronize();

    printBoard(newBoard);

    hipFree(oldBoard);
    hipFree(newBoard);

    return 0;
}