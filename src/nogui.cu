#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
extern "C" {
#include "include/gameOfLife.h"
}

using namespace std;

__device__ int mod(int a, int b)
{
    return (a + b) % b;
}

__device__ char nextState(char slice[9])
{
    if (slice[4] == RED_DYING || slice[4] == GREEN_DYING || slice[4] == BLUE_DYING)
        return DEAD;
    if (slice[4] == RED_ALIVE)
        return RED_DYING;
    if (slice[4] == GREEN_ALIVE)
        return GREEN_DYING;
    if (slice[4] == BLUE_ALIVE)
        return BLUE_DYING;
    
    int red, green, blue;
    red = green = blue = 0;

    for (int i = 0; i < 9; i++)
    {
        if (i == 4)
            continue;
        else if (slice[i] == RED_ALIVE)
            red++;
        else if (slice[i] == GREEN_ALIVE)
            green++;
        else if (slice[i] == BLUE_ALIVE)
            blue++;
    }

    if (red + green + blue == 4)
    {
        if (red >= 2)
            return RED_ALIVE;
        if (green >= 2)
            return GREEN_ALIVE;
        if (blue >= 2)
            return BLUE_ALIVE;
    }

    return DEAD;
}

__global__ void nextGeneration(char *oldBoard, char *newBoard)
{
    int x, y, left, right, up, down, stride = blockDim.x * gridDim.x;
    char slice[9];

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < WIDTH * HEIGHT; i += stride)
    {
        x = i % WIDTH;
        y = i - x;
        left = mod(x - 1, WIDTH);
        right = mod(x + 1, WIDTH);
        up = mod(y - WIDTH, WIDTH * HEIGHT);
        down = mod(y + WIDTH, WIDTH * HEIGHT);

        slice[0] = oldBoard[left + up];
        slice[1] = oldBoard[x + up];
        slice[2] = oldBoard[right + up];
        slice[3] = oldBoard[left + y];
        slice[4] = oldBoard[i];
        slice[5] = oldBoard[right + y];
        slice[6] = oldBoard[left + down];
        slice[7] = oldBoard[x + down];
        slice[8] = oldBoard[right + down];

        newBoard[i] = nextState(slice);
    }
}

void printBoard(char *board) {
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 16; j++) {
            switch(board[i * WIDTH + j])
            {
                case RED_ALIVE: cout << "R"; break;
                case RED_DYING: cout << "r"; break;
                case GREEN_ALIVE: cout << "G"; break;
                case GREEN_DYING: cout << "g"; break;
                case BLUE_ALIVE: cout << "B"; break;
                case BLUE_DYING: cout << "b"; break;
                case DEAD: cout << " "; break;
                default: cout << "X"; break;
            }
        }
        cout << endl;
    }
    cout << endl;
}

__global__ void initRand(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void cudaInitiateRandomBoard(hiprandState *globalState, char *board) {
    int stride = blockDim.x * gridDim.x, idx = blockIdx.x * blockDim.x + threadIdx.x;
    float random;
    hiprandState localState;

    for (int i = idx; i < WIDTH * HEIGHT; i += stride)
    {
        localState = globalState[idx % (blockDim.x * blockDim.x)];
        random = hiprand_uniform(&(localState));
        if (random <= 0.0833)
            board[i] = RED_ALIVE;
        else if (random <= 0.1666)
            board[i] = GREEN_ALIVE;
        else if (random <= 0.2500)
            board[i] = BLUE_ALIVE;
        else
            board[i] = DEAD;
        globalState[idx % (blockDim.x * blockDim.x)] = localState;
    }
}

int main (void) {
    char *oldBoard, *newBoard;
    hiprandState *devStates;
    int blockSize = 256;
    int numBlocks = (WIDTH * HEIGHT + blockSize - 1) / blockSize;

    hipMalloc(&devStates, blockSize * blockSize * sizeof(hiprandState));
    hipMallocManaged(&oldBoard, WIDTH * HEIGHT * sizeof(char));
    hipMallocManaged(&newBoard, WIDTH * HEIGHT * sizeof(char));
    hipDeviceSynchronize();

    initRand<<<blockSize, blockSize>>>(devStates, time(NULL));
    cudaInitiateRandomBoard<<<numBlocks, blockSize>>>(devStates, oldBoard);
    hipDeviceSynchronize();

    printBoard(oldBoard);

    nextGeneration<<<numBlocks, blockSize>>>(oldBoard, newBoard);
    hipDeviceSynchronize();

    printBoard(newBoard);

    hipFree(oldBoard);
    hipFree(newBoard);

    return 0;
}